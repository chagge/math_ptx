#include "hip/hip_runtime.h"
#include "types.h"

#define MATH_BASIC(fn, T)                       \
    __device__ T ___##fn(T a)                   \
    {                                           \
        return fn##f((float)a);                 \
    }                                           \


#define MATH(fn)                                \
    MATH_BASIC(fn, float)                       \
    MATH_BASIC(fn, int)                         \
    MATH_BASIC(fn, uint)                        \
    MATH_BASIC(fn, char)                        \
    MATH_BASIC(fn, uchar)                       \
    __device__ double ___##fn(double a)         \
    {                                           \
        return fn(a);                           \
    }                                           \


MATH(exp)
MATH(expm1)
MATH(erf)
MATH(erfc)

MATH(log)
MATH(log10)
MATH(log1p)

MATH(sqrt)
MATH(cbrt)

#define MATH2_BASIC(fn, T)                      \
    __device__ T ___##fn(T a, T b)              \
    {                                           \
        return fn##f((float)a, (float)b);       \
    }                                           \

#define MATH2(fn)                                   \
    MATH2_BASIC(fn, float)                          \
    MATH2_BASIC(fn, int)                            \
    MATH2_BASIC(fn, uint)                           \
    MATH2_BASIC(fn, char)                           \
    MATH2_BASIC(fn, uchar)                          \
    __device__ double ___##fn(double a, double b)   \
    {                                               \
        return fn(a, b);                            \
    }                                               \

MATH2(pow)
