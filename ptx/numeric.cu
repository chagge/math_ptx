#include "hip/hip_runtime.h"
#include "types.h"

#define MATH_BASIC(fn, T)                       \
    __device__ T ___##fn(T a)                   \
    {                                           \
        return fn##f((float)a);                 \
    }                                           \


#define MATH(fn)                                \
    MATH_BASIC(fn, float)                       \
    MATH_BASIC(fn, int)                         \
    MATH_BASIC(fn, uint)                        \
    MATH_BASIC(fn, char)                        \
    MATH_BASIC(fn, uchar)                       \
    __device__ double ___##fn(double a)         \
    {                                           \
        return fn(a);                           \
    }                                           \

MATH(floor)
MATH(ceil)
MATH(round)

MATH(tgamma)
MATH(lgamma)


#define fremainderf(a, b) remainderf((a), (b))
#define fremainder(a, b) remainder((a), (b))

#define MATH2_BASIC(fn, T)                      \
    __device__ T ___##fn(T a, T b)              \
    {                                           \
        return f##fn##f((float)a, (float)b);    \
    }                                           \

#define MATH2(fn)                                   \
    MATH2_BASIC(fn, float)                          \
    MATH2_BASIC(fn, int)                            \
    MATH2_BASIC(fn, uint)                           \
    MATH2_BASIC(fn, char)                           \
    MATH2_BASIC(fn, uchar)                          \
    __device__ double ___##fn(double a, double b)   \
    {                                               \
        return f##fn(a, b);                         \
    }                                               \

MATH2(min)
MATH2(max)
MATH2(mod)
MATH2(remainder)

#define COMPARE_CPLX(fn, op, T)                 \
    __device__ T ___##fn(T a, T b)              \
    {                                           \
        return cabs2(a) op cabs2(b) ? a : b;    \
    }                                           \

COMPARE_CPLX(min, <, cfloat)
COMPARE_CPLX(min, <, cdouble)
COMPARE_CPLX(max, >, cfloat)
COMPARE_CPLX(max, >, cdouble)
